//
// Created by Gary27 on 2025/7/3.
//

#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceID = 0;
    if (hipSuccess != hipGetDevice(&deviceID)) return -10;

    hipSharedMemConfig sharedMemConfig;
    hipDeviceGetSharedMemConfig(&sharedMemConfig);
    std::cout << "cudaSharedMemConfig: " << sharedMemConfig << std::endl;

}