
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <string>
#include <ctime>
#include <iostream>
//#include <cuda_runtime.h>

void initialData(float *ip, int size){
    time_t t;
    srand((unsigned int)time(&t));
    for(int i=0;i<size;++i){
        ip[i] = (float)( rand() & 0xFF ) / 10.0F;
    }
}

__global__ void sumArrayOnDevice(float *A, float *B, float *C, const int N){

    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];

}

int main(){

    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);

    auto h_A = (float *) malloc(nBytes);
    auto h_B = (float *) malloc(nBytes);
    auto h_C = (float *) malloc(nBytes);
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    sumArrayOnDevice<<<1, nElem>>>(d_A, d_B, d_C, nElem);

    hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);

    for(int i=0;i<nElem;++i){
        std::cout << "i: " << h_C[i] << "\n";
    }

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}