//
// Created by Gary27 on 2025/3/28.
//

#include <hip/hip_runtime.h>
#include <iostream>

int main(){
  hipStream_t default_stream = hipStreamDefault;  // 获取默认流
  unsigned int flags;
  if (hipStreamGetFlags(default_stream, &flags) != hipSuccess) {
    std::cerr << "Failed to get default_stream flags" << std::endl;
    hipStreamDestroy(default_stream);
    return 1;
  }

  std::cout << "Stream flags: " << flags << std::endl;
  // if (flags == cudaStreamDefault) {
  //   std::cout << "This is a default stream" << std::endl;
  // } else if (flags == hipStreamNonBlocking) {
  //   std::cout << "This is a non-blocking stream" << std::endl;
  // }



  return 0;
}