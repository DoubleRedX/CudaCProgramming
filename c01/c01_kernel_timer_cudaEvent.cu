#include "hip/hip_runtime.h"
//
// Created by Gary27 on 2024/12/26.
//

#include <chrono>
#include <iostream>
#include <array>
#include <hip/hip_runtime.h>
#include "aux.h"
#include "helper.h"

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    C[idx] = A[idx] + B[idx];
  }
}

int main(){

  const int N = 1024;
  constexpr int array_bytes_size = N * sizeof(float);
  std::array<float, N> ha, hb, hc;
  hipEvent_t t0, t1, t2, t3;
  CUDA_CHECK(hipEventCreate(&t0));
  CUDA_CHECK(hipEventCreate(&t1));
  CUDA_CHECK(hipEventCreate(&t2));
  CUDA_CHECK(hipEventCreate(&t3));


  initializeData<float, N>(ha, 1.);
  initializeData<float, N>(hb, 1.);
  initializeData<float, N>(hc, 0.);

  float *da, *db, *dc;

  const int block_size = 256;
  const int grid_size = (N + block_size - 1) / block_size;

  CUDA_CHECK(hipEventRecord(t0));
  CUDA_CHECK(hipMalloc(&da, array_bytes_size));
  CUDA_CHECK(hipMalloc(&db, array_bytes_size));
  CUDA_CHECK(hipMalloc(&dc, array_bytes_size));
  CUDA_CHECK(hipMemcpy(da, ha.data(), array_bytes_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(db, hb.data(), array_bytes_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dc, hc.data(), array_bytes_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipEventRecord(t1));
  vectorAdd<<<grid_size, block_size>>>(da, db, dc, N);
  CUDA_CHECK(hipEventRecord(t2));
  CUDA_CHECK(hipMemcpy(hc.data(), dc, array_bytes_size, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipEventRecord(t3));
  CUDA_CHECK(hipFree(da));
  CUDA_CHECK(hipFree(db));
  CUDA_CHECK(hipFree(dc));
  float t_io_h2d = 0;
  float t_kernel_exec = 0;
  float t_io_d2h = 0;
  float t_total = 0;
  hipEventElapsedTime(&t_io_h2d, t0, t1);
  hipEventElapsedTime(&t_kernel_exec, t1, t2);
  hipEventElapsedTime(&t_io_d2h, t2, t3);
  hipEventElapsedTime(&t_total, t0, t3);
  std::cout << "t_io_h2d time (CUDA events): " << t_io_h2d << " ms" << std::endl;
  std::cout << "Kernel execution time (CUDA events): " << t_kernel_exec << " ms" << std::endl;
  std::cout << "t_io_d2h time (CUDA events): " << t_io_d2h << " ms" << std::endl;
  std::cout << "t_total time (CUDA events): " << t_total << " ms" << std::endl;
  return 0;
}

/*
t_io_h2d time (CUDA events): 0.139264 ms
Kernel execution time (CUDA events): 0.077824 ms
t_io_d2h time (CUDA events): 0.014464 ms
t_total time (CUDA events): 0.231552 ms
 */