//
// Created by Gary27 on 2025/1/20.
//

#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>

__device__ uint64_t shr(uint64_t value, uint32_t shift){
    // Shift a right by the amount specified by unsigned 32-bit value in b.
    uint64_t res;
    asm("shr.b64 %0, %1, %2;" : "=l"(res) : "l"(value), "r"(shift));
    return res;
}

__global__ void test_shr(){
    uint64_t value = 0x12345678FFFFFFFF;
    uint64_t res;
    uint32_t shift = 4 * 2;
    res = shr(value, 64 - shift);

    printf("original value: %lX\n", value);
    printf("after shl value: %lX\n", res);
}


int main(){
    test_shr<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}