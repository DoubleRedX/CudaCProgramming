//
// Created by Gary27 on 2025/1/17.
//
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

__global__ void fma_example(double* result) {
    double a = 1.523224234652346235462242443;
    double b = 2.52224242422562356256443;
    double c = 3.54242422562562562564423;

    result[0] = __fma_rn(a, b, c); // 舍入到最接近的偶数
    result[1] = __fma_rd(a, b, c); // 向负无穷方向舍入
    result[2] = __fma_ru(a, b, c); // 向正无穷方向舍入
    result[3] = __fma_rz(a, b, c); // 向零方向舍入
}

int main() {
    const int num_results = 4;
    double h_result[num_results];
    double* d_result;

    // 分配设备内存
    hipMalloc(&d_result, num_results * sizeof(double));

    // 启动内核
    fma_example<<<1, 1>>>(d_result);

    // 将结果拷贝回主机
    hipMemcpy(h_result, d_result, num_results * sizeof(double), hipMemcpyDeviceToHost);

    // 打印结果
    std::cout << std::fixed << std::setprecision(50);
    std::cout << "__fma_rn: " << h_result[0] << std::endl; // 舍入到最接近的偶数
    std::cout << "__fma_rd: " << h_result[1] << std::endl; // 向负无穷方向舍入
    std::cout << "__fma_ru: " << h_result[2] << std::endl; // 向正无穷方向舍入
    std::cout << "__fma_rz: " << h_result[3] << std::endl; // 向零方向舍入

    // 释放设备内存
    hipFree(d_result);

    return 0;
}