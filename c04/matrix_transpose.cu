//
// Created by Gary27 on 2025/3/7.
//

#include <hip/hip_runtime.h>

#include <cstdio>

template<typename T>
__global__ void matrix_transpose_row(T *in, T *out, int m, int n){
    auto i = threadIdx.y + blockIdx.y * blockDim.y;
    auto j = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < m && j < n){
        out[i + j * m] = in[j + i * n];  // read i, j --> write j, i
    }
}
//
//template<typename T>
//__global__ void matrix_transpose_col(T *in, T *out, int m, int n){
//    auto i = threadIdx.y + blockDim.y * blockIdx.y;
//    auto j = threadIdx.x + blockDim.x * blockIdx.x;
//    if(i < m && j < n){
//        out[] = in[];
//    }
//}


int main(int argc, char** argv){


    const int originalHeight = 3;
    const int originalWidth = 4;

    const int size = originalWidth * originalHeight * sizeof(float);

    float h_input[] = {1.0f, 2.0f, 3.0f, 4.0f,
                       5.0f, 6.0f, 7.0f, 8.0f,
                       9.0f, 10.0f, 11.0f, 12.0f};
    float h_output[originalWidth * originalHeight];

    float *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize(
            (originalWidth + blockSize.x - 1) / blockSize.x,
            (originalHeight + blockSize.y - 1) / blockSize.y
    );
    matrix_transpose_row<<<gridSize, blockSize>>>(d_input, d_output, originalHeight, originalWidth);

    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < originalWidth; ++i) {
        for (int j = 0; j < originalHeight; ++j) {
            printf("%.0f ", h_output[i * originalWidth + j]);
        }
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_output);







    return 0;
}