#include "hip/hip_runtime.h"
//
// Created by Gary27 on 2025/2/11.
//

#include <hip/hip_runtime.h>
#include <cstdio>
#include "spdlog/spdlog.h"

__device__ float devData = 1.0f;

__global__ void checkGlobalVariable(){
    printf("Device: devData is %f\n", devData);
    devData += 2.0f;
}

int main(){
    float value = 10.;
    hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float ));
    checkGlobalVariable<<<1,1>>>();
    // ---------------核心代码------------------------
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    // ---------------核心代码------------------------
//    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float ));
    printf("Host: value is %f\n", value);
    printf("Host: devData is %f\n", devData);
//    spdlog::info("Host: Value is {}\n", value);
    hipDeviceReset();
    return 0;
}