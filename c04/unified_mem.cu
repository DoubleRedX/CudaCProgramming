//
// Created by Gary27 on 2025/2/11.
//
#include <hip/hip_runtime.h>

#include "spdlog/spdlog.h"

int main(){


    float* value;
    hipMallocManaged(&value, sizeof(float));
    spdlog::info("value is {}", *value);
    *value = 100.0;
    spdlog::info("value is {}", *value);
    *value = 120.0;
    spdlog::info("value is {}", *value);
    *value += 100.0;
    spdlog::info("value is {}", *value);

    hipFree(value);

    return 0;
}