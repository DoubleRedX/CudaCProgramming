//
// Created by Gary27 on 2025/2/11.
//

// memTransfer

#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

int main(){
    int dev = 1;
    hipSetDevice(dev);
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, dev);

    unsigned int data_size = 1<<28;
    auto n_bytes = data_size * sizeof(float );

    std::cout << "Device " << dev << ": " << prop.name << " memory size " << data_size << " nbyte " << (n_bytes/1024.0f/1024.0f) << "MB\n";

    // allocate host mem
    auto *hst_mem = new float[data_size];
    // allocate device mem
    float *da;
    hipMalloc(&da, n_bytes);
    // initialize host mem
    std::fill_n(hst_mem, data_size, 0.5f);  // or for(int i=0;i<int_size;++i) h_mem[i] = 0.5f;


    // allocate device mem
    float* dev_mem;
    hipMalloc(&dev_mem, n_bytes);

    // transfer data from host to device
    hipMemcpy(dev_mem, hst_mem, data_size, hipMemcpyHostToDevice);

    // transfer data from device to host
    hipMemcpy(hst_mem, dev_mem, data_size, hipMemcpyDeviceToHost);

    // free mem
    delete[] hst_mem;
    hipFree(dev_mem);
    hipDeviceReset();
    return 0;
}